
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 256

__global__ void saxpy(int N, float *x, float *y, float *z)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    for(int k = 0; k < 1000; ++k) {
      z[i] = x[i] + y[i];
      z[i] += 1;
    }
  }
}

int main(void)
{
  clock_t start_time = clock();
  int N = 1<<20;
  float *x, *y, *z, *d_x, *d_y, *d_z;
  int sum = 0;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  z = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+THREAD_PER_BLOCK-1)/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(N, d_x, d_y, d_z);

  hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);
  clock_t end_time = clock();
  float time_c = 1000*(((float)(end_time - start_time))/CLOCKS_PER_SEC);
  // time_c in milliseconds
  printf("Time consumed: %f millsec \n", time_c);

  for (int i = 0; i < N; ++i) {
    sum += z[i];
  }
  printf("Sum: %d \n", sum);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  free(x);
  free(y);
  free(z);
}
